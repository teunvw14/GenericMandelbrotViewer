#include "hip/hip_runtime.h"
#include <math.h>

// CUDA imports
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <>

#include "util/color_palette.h"
#include "mandelbrot_image.h"
#include "constants.h"


// Create a grid of complex numbers around the center point (center_real, center_imag).
__global__ void build_complex_grid_cuda(mandelbrot_image* image)
{
    int block_index_x = blockIdx.x;
    int block_stride_x = gridDim.x;
    int thread_index_x = threadIdx.x;
    int thread_stride_x = blockDim.x;

    double step_x = 2 * image->draw_radius_x / image->resolution_x;
    double step_y = 2 * image->draw_radius_y / image->resolution_y;
    double point_re;
    double point_im;
    int index;
    // Start drawing in the bottom left, go row by row.
    for (int pixel_y = block_index_x; pixel_y < image->resolution_y; pixel_y += block_stride_x)	{
        point_im = image->center_imag + pixel_y * step_y - image->draw_radius_y;
        for (int pixel_x = thread_index_x; pixel_x < image->resolution_x; pixel_x += thread_stride_x) {
            index = pixel_y * image->resolution_x + pixel_x;
            point_re = image->center_real + pixel_x * step_x - image->draw_radius_x;
            (image->points)[index] = make_hipDoubleComplex(point_re, point_im);
            (image->iterated_points)[index] = make_hipDoubleComplex(point_re, point_im);
        }
    }
}

extern "C" void launch_build_complex_grid_cuda(int num_blocks, int block_size, mandelbrot_image* image)
{
    build_complex_grid_cuda <<< num_blocks, block_size >>> (image);
}


__global__ void reset_render_arrays_cuda(mandelbrot_image* image)
{
    int block_index_x = blockIdx.x;
    int block_stride_x = gridDim.x;
    int thread_index_x = threadIdx.x;
    int thread_stride_x = blockDim.x;
    int index;
    // Start drawing in the bottom left, go row by row.
    for (int pixel_y = block_index_x; pixel_y < image->resolution_y; pixel_y += block_stride_x) {
        for (int pixel_x = thread_index_x; pixel_x < image->resolution_x; pixel_x += thread_stride_x) {
            index = pixel_y * image->resolution_x + pixel_x;
            (image->iterationsArr)[index] = 0;
            (image->squared_absolute_values)[index] = 0;
        }
    }
}

extern "C" void launch_reset_render_arrays_cuda(int num_blocks, int block_size, mandelbrot_image* image)
{
    reset_render_arrays_cuda <<< num_blocks, block_size >>> (image);
}


__global__ void mandelbrot_iterate_cuda(mandelbrot_image* image)
{
    int block_index_x = blockIdx.x;
    int block_stride_x = gridDim.x;

    int thread_index_x = threadIdx.x;
    int thread_stride_x = blockDim.x;
    int index;
    unsigned int iterations_;

    for (int pixel_y = block_index_x; pixel_y < image->resolution_y; pixel_y += block_stride_x) {
        for (int pixel_x = thread_index_x; pixel_x < image->resolution_x; pixel_x += thread_stride_x) {
            // Calculate the iterations required for a given point to exceed the escape radius.
            index = pixel_y * image->resolution_x + pixel_x;
            hipDoubleComplex starting_number = (image->points)[index];
            hipDoubleComplex iterated_point = (image->iterated_points)[index];
            double sq_abs = (image->squared_absolute_values)[index];
            iterations_ = (image->iterationsArr)[index];
            while (iterations_ < image->max_iterations && sq_abs < image->escape_radius_squared) {
                iterated_point = make_hipDoubleComplex(iterated_point.x * iterated_point.x - iterated_point.y * iterated_point.y + starting_number.x,
                                                      2 * iterated_point.x * iterated_point.y + starting_number.y);
                sq_abs = iterated_point.x * iterated_point.x + iterated_point.y * iterated_point.y;
                iterations_++;
            }
            (image->iterated_points)[index] = iterated_point;
            (image->iterationsArr)[index] = iterations_;
            (image->squared_absolute_values)[index] = sq_abs;
        }
    }
}

extern "C" void launch_mandelbrot_iterate_cuda(int num_blocks, int block_size, mandelbrot_image* image)
{
    mandelbrot_iterate_cuda <<< num_blocks, block_size >>> (image);
}


__global__ void color_cuda(mandelbrot_image* image, int coloring_mode)
{
    // Do some coloring!

    int block_index_x = blockIdx.x;
    int block_stride_x = gridDim.x;

    int thread_index_x = threadIdx.x;
    int thread_stride_x = blockDim.x;
    int index;
    unsigned int iterations;

    for (int pixel_y = block_index_x; pixel_y < image->resolution_y; pixel_y += block_stride_x) {
        for (int pixel_x = thread_index_x; pixel_x < image->resolution_x; pixel_x += thread_stride_x) {
            // Calculate the iterations required for a given point to exceed the escape radius.
            index = pixel_y * image->resolution_x + pixel_x;
            iterations = (image->iterationsArr)[index];
            if (iterations == image->max_iterations) {
                // Values that don't escape are colored black:
                (image->pixels_rgb)[3 * index + 0] = 0; // Red value
                (image->pixels_rgb)[3 * index + 1] = 0; // Green value
                (image->pixels_rgb)[3 * index + 2] = 0; // Blue value
            }
            else {
                // Calculate the iterations required for a given point to exceed the escape radius.
            // Calculate the iterations required for a given point to exceed the escape radius.
                index = pixel_y * image->resolution_x + pixel_x;
                iterations = (image->iterationsArr)[index];
                if (iterations == image->max_iterations) {
                    // Values that don't escape are colored black:
                    (image->pixels_rgb)[3 * index + 0] = 0; // Red value
                    (image->pixels_rgb)[3 * index + 1] = 0; // Green value
                    (image->pixels_rgb)[3 * index + 2] = 0; // Blue value
                }
                else {
                    color_rgb pixel_color;
                    // TODO: fix coloring modes in CUDA
                    /*if (coloring_mode == COLORING_SIMPLE) {
                        simple_palette hacker_green_palette;
                        hacker_green_palette.start_color = black;
                        hacker_green_palette.end_color = white;
                        float factor = sqrtf((float)iterations / (float)image->max_iterations);
                        pixel_color = lerp_color(hacker_green_palette.start_color, hacker_green_palette.end_color, factor);
                    }*/
                    /*if (coloring_mode == COLORING_PALETTE) {
                        palette p = palette_pretty;
                        pixel_color = p.colors[iterations % p.length];
                    }*/
                    if (coloring_mode == COLORING_SIMPLE || coloring_mode == COLORING_PALETTE || coloring_mode == COLORING_SMOOTH) {
                        float f_iterations = (float)iterations;
                        float f_max_iterations = (float)image->max_iterations;
                        // Smooth colors!
                        float escape_size = __double2float_rn(image->squared_absolute_values[index]);
                        float smoothed_iterations = iterations + 1 - log2f(log(escape_size)) + sqrtf(sqrtf(image->draw_radius_x));
                        float H = 360 * smoothed_iterations / f_max_iterations;
                        float S = .65;
                        float V = 1;

                        // HSV to RGB conversion, yay!
                        // TODO: look into edge cases for H and why they happen.
                        //if (H > 360 || H < 0 || S > 1 || S < 0 || V > 1 || V < 0)
                        //{
                        //printf("The given HSV values are not in valid range.\n H: %f S: %.2f, V: %.2f\n", H, S, V);
                        //printf("Iterations: %f\n", f_iterations);
                        //}
                        float h = H / 60;
                        float C = S * V;
                        float X = C * (1 - fabsf((fmodf(h, 2) - 1)));
                        float m = V - C;
                        float r, g, b;
                        if (h >= 0 && h <= 1) {
                            r = C;
                            g = X;
                            b = 0;
                        }
                        else if (h > 1 && h < 2) {
                            r = X;
                            g = C;
                            b = 0;
                        }
                        else if (h > 2 && h <= 3) {
                            r = 0;
                            g = C;
                            b = X;
                        }
                        else if (h > 3 && h <= 4) {
                            r = 0;
                            g = X;
                            b = C;
                        }
                        else if (h > 4 && h <= 5) {
                            r = X;
                            g = 0;
                            b = C;
                        }
                        else if (h > 5 && h <= 6) {
                            r = C;
                            g = 0;
                            b = X;
                        }
                        else { // color white to make stand out
                            r = 1 - m;
                            g = 1 - m;
                            b = 1 - m;
                        }
                        unsigned char red = (r + m) * 255;
                        unsigned char green = (g + m) * 255;
                        unsigned char blue = (b + m) * 255;
                        // End of conversion.

                        // Cap RGB values to 255
                        if (red > 255) {
                            red = 255;
                        }
                        if (green > 255) {
                            green = 255;
                        }
                        if (blue > 255) {
                            blue = 255;
                        }
                        pixel_color.r = red;
                        pixel_color.g = green;
                        pixel_color.b = blue;
                    }
                    (image->pixels_rgb)[3 * index + 0] = pixel_color.r; // Red value
                    (image->pixels_rgb)[3 * index + 1] = pixel_color.g; // Green value
                    (image->pixels_rgb)[3 * index + 2] = pixel_color.b; // Blue value
                }
            }
        }
    }
}

extern "C" void launch_color_cuda(int num_blocks, int block_size, mandelbrot_image* image, int coloring_mode)
{
    color_cuda <<< num_blocks, block_size >>> (image, coloring_mode);
}
