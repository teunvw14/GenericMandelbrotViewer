#include "hip/hip_runtime.h"
#include <glfw3.h>
#include <stdio.h>
#include <math.h>

// CUDA imports
#include <hip/hip_runtime.h>
#include <>

// Debugging:
#include <windows.h>
bool debugging_enabled = false;

// Define starting parameters for the mandelbrot
double center_x = 0.0;
double center_y = 0.0;
int resolution_x = 128;
int resolution_y = 128;
double draw_radius = 2.5;
double escape_radius_squared = 4; // escape_radius = 2^7 = 256
int max_iterations = 64;

bool incremental_iteration = false;
int iterations_per_frame; // value set in main()
int incremental_iterations_per_frame = 4;
int rendered_iterations = 0;

// Cuda parameters:
int cuda_block_size = 256;
int cuda_num_blocks = int(ceil(resolution_x * resolution_y / cuda_block_size));
bool cuda_device_available = false;

// Define variables used to imaginary number values for each pixel
double* points_real;
double* points_imag;
double* iterated_points_real;
double* iterated_points_imag;

double* squared_absolute_values;
unsigned char* pixels_rgb;
unsigned int* iterationsArr;


__global__ void build_complex_grid_cuda(
    double center_x, double center_y, 
    double draw_radius, 
    int resolution_x, int resolution_y, 
    double* points_real,
    double* points_imag,
    double* iterated_points_real,
    double* iterated_points_imag
    )
{
    // Create a grid of complex numbers around the center point (center_x, center_y).
    
    int thread_index = threadIdx.x;
	int thread_stride = blockDim.x;

	double step_y = 2 * draw_radius / resolution_y;
	double step_x = 2 * draw_radius / resolution_x;
	double point_re;
	double point_im;
	int index;
	// Start drawing in the bottom left, go row by row.
	for (int pixel_y = thread_index; pixel_y < resolution_y; pixel_y += thread_stride)
	{
		point_im = center_y + pixel_y * step_y - draw_radius;
		for (int pixel_x = 0; pixel_x < resolution_x; pixel_x++)
		{
			index = pixel_y * resolution_y + pixel_x;
			point_re = center_x + pixel_x * step_x - draw_radius;
            points_real[index] = point_re;
            points_imag[index] = point_im;
            iterated_points_real[index] = point_re;
            iterated_points_imag[index] = point_im;
		}
	}
}

void build_complex_grid_non_cuda(
        double center_x, double center_y,
        double draw_radius,
        int resolution_x, int resolution_y,
        double* points_real,
        double* points_imag,
        double* iterated_points_real,
        double* iterated_points_imag
    )
{
    // Create a grid of complex numbers around the center point (center_x, center_y).
    double step_y = 2 * draw_radius / resolution_y;
    double step_x = 2 * draw_radius / resolution_x;
    double point_re;
    double point_im;
    int index;
    // Start drawing in the bottom left, go row by row.
    for (int pixel_y = 0; pixel_y < resolution_y; pixel_y++)
    {
        point_im = center_y + pixel_y * step_y - draw_radius;
        for (int pixel_x = 0; pixel_x < resolution_x; pixel_x++)
        {
            index = pixel_y * resolution_y + pixel_x;
            point_re = center_x + pixel_x * step_x - draw_radius;
            points_real[index] = point_re;
            points_imag[index] = point_im;
            iterated_points_real[index] = point_re;
            iterated_points_imag[index] = point_im;
        }
    }
}

__global__ void mandelbrot_iterate_cuda(
    int max_iterations,
    double escape_radius_squared,
    int resolution_x, int resolution_y,
    double* points_real,
    double* points_imag,
    double* iterated_points_real,
    double* iterated_points_imag,
    double* squared_absolute_values,
    unsigned int* iterationsArr
)
{
    int block_index_x = blockIdx.x;
    int block_stride_x = gridDim.x;
    //int block_index_y = blockIdx.y;
    //int block_stride_y = gridDim.y;

    int thread_index_x = threadIdx.x;
    int thread_stride_x = blockDim.x;
    //int thread_index_y = threadIdx.y;
    //int thread_stride_y = blockDim.y;
    int index;

    //printf("thread_index_x: %i | block_index_x: %i | thread_stride_x: %i | block_stride_x: %i\n", thread_index_x, block_index_x, thread_stride_x, block_stride_x);
    for (int pixel_y = block_index_x; pixel_y < resolution_y; pixel_y += block_stride_x)
    {
        for (int pixel_x = thread_index_x; pixel_x < resolution_x; pixel_x += thread_stride_x)
        {
            // Calculate the iterations required for a given point to exceed the escape radius.
            index = pixel_y * resolution_y + pixel_x;
            double c_real = points_real[index];
            double c_imag = points_imag[index];
            double it_point_real = iterated_points_real[index];
            double it_point_imag = iterated_points_imag[index];
            double sq_abs = squared_absolute_values[index];
            unsigned int iterations_ = iterationsArr[index];
            while (iterations_ < max_iterations && sq_abs < escape_radius_squared) {
                it_point_real = it_point_real * it_point_real - it_point_imag * it_point_imag + c_real;
                it_point_imag = 2 * it_point_real * it_point_imag + c_imag;
                sq_abs = it_point_real * it_point_real + it_point_imag * it_point_imag;
                iterations_++;
            }
            iterated_points_real[index] = it_point_real;
            iterated_points_imag[index] = it_point_imag;
            iterationsArr[index] = iterations_;
            squared_absolute_values[index] = sq_abs;
        }
    }
}

void mandelbrot_iterate_non_cuda(
    int max_iterations,
    double escape_radius_squared,
    int resolution_x, int resolution_y,
    double* points_real,
    double* points_imag,
    double* iterated_points_real,
    double* iterated_points_imag,
    double* squared_absolute_values,
    unsigned int* iterationsArr
)
{
    int index = 0;

    for (int pixel_y = 0; pixel_y < resolution_y; pixel_y++)
    {
        for (int pixel_x = 0; pixel_x < resolution_x; pixel_x++)
        {
            // Calculate the iterations required for a given point to exceed the escape radius.
            double c_real = points_real[index];
            double c_imag = points_imag[index];
            double it_point_real = iterated_points_real[index];
            double it_point_imag = iterated_points_imag[index];
            double sq_abs = squared_absolute_values[index];
            unsigned int iterations_ = iterationsArr[index];
            while (iterations_ < max_iterations && sq_abs < escape_radius_squared) {
                it_point_real = it_point_real * it_point_real - it_point_imag * it_point_imag + c_real;
                it_point_imag = 2 * it_point_real * it_point_imag + c_imag;
                sq_abs = it_point_real * it_point_real + it_point_imag * it_point_imag;
                iterations_++;
            }
            iterated_points_real[index] = it_point_real;
            iterated_points_imag[index] = it_point_imag;
            iterationsArr[index] = iterations_;
            squared_absolute_values[index] = sq_abs;
            index++;
        }
    }
}


__global__ void color_cuda(
    int max_iterations,
    unsigned int* iterationsArr,
    double * squared_absolute_values,
    int resolution_x,
    int resolution_y,
    double draw_radius,
    unsigned char * rgb_data
)
{
    // Do some coloring!

    int block_index_x = blockIdx.x;
    int block_stride_x = gridDim.x;
    //int block_index_y = blockIdx.y;
    //int block_stride_y = gridDim.y;

    int thread_index_x = threadIdx.x;
    int thread_stride_x = blockDim.x;
    //int thread_index_y = threadIdx.y;
    //int thread_stride_y = blockDim.y;
    int index;
    unsigned int iterations;

    //printf("thread_index_x: %i | block_index_x: %i | thread_stride_x: %i | block_stride_x: %i\n", thread_index_x, block_index_x, thread_stride_x, block_stride_x);
    for (int pixel_y = block_index_x; pixel_y < resolution_y; pixel_y += block_stride_x)
    {
        for (int pixel_x = thread_index_x; pixel_x < resolution_x; pixel_x += thread_stride_x)
        {
            // Calculate the iterations required for a given point to exceed the escape radius.
            index = pixel_y * resolution_y + pixel_x;
            iterations = iterationsArr[index];
            if (iterations == max_iterations)
            {
                // Values that don't escape are colored black:
                rgb_data[3 * index + 0] = 25; // Red value
                rgb_data[3 * index + 1] = 25; // Green value
                rgb_data[3 * index + 2] = 25; // Blue value
            } 
            else
            {
                float f_iterations = (float)iterations;
                float f_max_iterations = (float)max_iterations;
                // Smooth colors!
                float escape_size = __double2float_rn(squared_absolute_values[index]);
                float smoothed_iterations = iterations + 1 - log2f(log(escape_size)) + sqrtf(sqrtf(draw_radius));
                float H = 360*smoothed_iterations / f_max_iterations;
                float S = .65;
                float V = 1;


                // HSV to RGB conversion, yay!
                // TODO: look into edge cases for H and why they happen.
                //if (H > 360 || H < 0 || S > 1 || S < 0 || V > 1 || V < 0)
                //{
                    //printf("x");
                    //printf("The given HSV values are not in valid range.\n H: %f S: %.2f, V: %.2f\n", H, S, V);
                    //printf("Iterations: %f\n", f_iterations);
                //}
                float h = H / 60;
                float C = S * V;
                float X = C * (1 - fabsf((fmodf(h, 2) - 1)));
                float m = V - C;
                float r, g, b;
                if (h >= 0 && h <= 1)
                {
                    r = C;
                    g = X;
                    b = 0;
                }
                else if (h > 1 && h < 2)
                {
                    r = X;
                    g = C;
                    b = 0;
                }
                else if (h > 2 && h <= 3)
                {
                    r = 0;
                    g = C;
                    b = X;
                }
                else if (h > 3 && h <= 4)
                {
                    r = 0;
                    g = X;
                    b = C;
                }
                else if (h > 4 && h <= 5)
                {
                    r = X;
                    g = 0;
                    b = C;
                }
                else if (h > 5 && h <= 6)
                {
                    r = C;
                    g = 0;
                    b = X;
                }
                else // color white to make stand out
                {
                    r = 1-m;
                    g = 1-m;
                    b = 1-m;
                }
                unsigned char red = (r + m) * 255;
                unsigned char green = (g + m) * 255;
                unsigned char blue = (b + m) * 255;
                // End of conversion.

                // Cap RGB values to 255
                if (red > 255) { red = 255; }
                if (green > 255) { green = 255; }
                if (blue > 255) { blue = 255; }

                rgb_data[3 * index + 0] = red; // Red value
                rgb_data[3 * index + 1] = green; // Green value
                rgb_data[3 * index + 2] = blue; // Blue value
            }
        }
    }
}


void color_non_cuda(
    int max_iterations,
    unsigned int* iterationsArr,
    double* squared_absolute_values,
    int resolution_x,
    int resolution_y,
    double draw_radius,
    unsigned char* rgb_data
)
{
    // Do some coloring!
    int index;
    unsigned int iterations;

    //printf("thread_index_x: %i | block_index_x: %i | thread_stride_x: %i | block_stride_x: %i\n", thread_index_x, block_index_x, thread_stride_x, block_stride_x);
    for (int pixel_y = 0; pixel_y < resolution_y; pixel_y++)
    {
        for (int pixel_x = 0; pixel_x < resolution_x; pixel_x++)
        {
            // Calculate the iterations required for a given point to exceed the escape radius.
            index = pixel_y * resolution_y + pixel_x;
            iterations = iterationsArr[index];
            if (iterations == max_iterations)
            {
                // Values that don't escape are colored black:
                rgb_data[3 * index + 0] = 25; // Red value
                rgb_data[3 * index + 1] = 25; // Green value
                rgb_data[3 * index + 2] = 25; // Blue value
            }
            else
            {
                float f_iterations = (float)iterations;
                float f_max_iterations = (float)max_iterations;
                // Smooth colors!
                float escape_size = (float )(squared_absolute_values[index]);
                float smoothed_iterations = iterations + 1 - log2f(log(escape_size)) + sqrtf(sqrtf(draw_radius));
                float H = 360 * smoothed_iterations / f_max_iterations;
                float S = .65;
                float V = 1;


#pragma region HSV_to_RGB_Conversion
                // HSV to RGB conversion, yay!
                // TODO: look into edge cases for H and why they happen.
                //if (H > 360 || H < 0 || S > 1 || S < 0 || V > 1 || V < 0)
                //{
                    //printf("x");
                    //printf("The given HSV values are not in valid range.\n H: %f S: %.2f, V: %.2f\n", H, S, V);
                    //printf("Iterations: %f\n", f_iterations);
                //}
                float h = H / 60;
                float C = S * V;
                float X = C * (1 - fabsf((fmodf(h, 2) - 1)));
                float m = V - C;
                float r, g, b;
                if (h >= 0 && h <= 1)
                {
                    r = C;
                    g = X;
                    b = 0;
                }
                else if (h > 1 && h < 2)
                {
                    r = X;
                    g = C;
                    b = 0;
                }
                else if (h > 2 && h <= 3)
                {
                    r = 0;
                    g = C;
                    b = X;
                }
                else if (h > 3 && h <= 4)
                {
                    r = 0;
                    g = X;
                    b = C;
                }
                else if (h > 4 && h <= 5)
                {
                    r = X;
                    g = 0;
                    b = C;
                }
                else if (h > 5 && h <= 6)
                {
                    r = C;
                    g = 0;
                    b = X;
                }
                else // color white to make stand out
                {
                    r = 1 - m;
                    g = 1 - m;
                    b = 1 - m;
                }
                unsigned char red = (r + m) * 255;
                unsigned char green = (g + m) * 255;
                unsigned char blue = (b + m) * 255;
                // End of conversion.
#pragma endregion

                // Cap RGB values to 255
                if (red > 255) { red = 255; }
                if (green > 255) { green = 255; }
                if (blue > 255) { blue = 255; }

                rgb_data[3 * index + 0] = red; // Red value
                rgb_data[3 * index + 1] = green; // Green value
                rgb_data[3 * index + 2] = blue; // Blue value
            }
        }
    }
}

// GLFW
GLFWwindow* window;

void build_complex_grid()
{
    if (cuda_device_available) {
        build_complex_grid_cuda <<< 1, 1024 >>> (center_x, center_y, draw_radius, resolution_x, resolution_y, points_real, points_imag, iterated_points_real, iterated_points_imag);
        hipDeviceSynchronize();
    }
    else if (!(cuda_device_available)){
        build_complex_grid_non_cuda(center_x, center_y, draw_radius, resolution_x, resolution_y, points_real, points_imag, iterated_points_real, iterated_points_imag);
    }
}

void mandelbrot_iterate_and_color()
{
    if (cuda_device_available) {
        mandelbrot_iterate_cuda <<< cuda_num_blocks, cuda_block_size
            >>> (
                max_iterations,
                escape_radius_squared,
                resolution_x, resolution_y,
                points_real,
                points_imag,
                iterated_points_real,
                iterated_points_imag,
                squared_absolute_values,
                iterationsArr
            );

        hipDeviceSynchronize();
        color_cuda <<< cuda_num_blocks, cuda_block_size
        >>> (
                max_iterations,
                iterationsArr,
                squared_absolute_values,
                resolution_x,
                resolution_y,
                draw_radius,
                pixels_rgb
            );
        hipDeviceSynchronize();
    }
    else if (!(cuda_device_available)) {
        mandelbrot_iterate_non_cuda(
                max_iterations,
                escape_radius_squared,
                resolution_x, resolution_y,
                points_real,
                points_imag,
                iterated_points_real,
                iterated_points_imag,
                squared_absolute_values,
                iterationsArr
            );

        color_non_cuda(
                max_iterations,
                iterationsArr,
                squared_absolute_values,
                resolution_x,
                resolution_y,
                draw_radius,
                pixels_rgb
            );
    }
}

// Under maintenance.
void mandelbrot_iterate_n_and_color(int iterations)
{
    mandelbrot_iterate_and_color();
    //mandelbrot_iterate_and_color_cuda << < cuda_num_blocks, cuda_block_size >> > (iterations, escape_radius_squared, resolution_x, resolution_y, points, iterated_points, squared_absolute_values, pixels_rgb);
}


void reset_render_objects()
{
    // This function resets all the variables that are used for rendering the Mandelbrot. 


    // Reset the `squared_absolute_values` to zero by allocating the memory space again.
    if (cuda_device_available) {
        hipFree(squared_absolute_values);
        hipFree(iterationsArr);
        hipMallocManaged(&squared_absolute_values, resolution_x * resolution_y * sizeof(double));
        hipMallocManaged(&iterationsArr, resolution_x * resolution_y * sizeof(unsigned int));
        // Synchronize the GPU so the whole thing doesn't crash.
        hipDeviceSynchronize();
    }
    else if (!(cuda_device_available)) {
        free(squared_absolute_values);
        free(iterationsArr);
        squared_absolute_values = (double*)malloc(resolution_x * resolution_y * sizeof(double));
        iterationsArr = (unsigned int*)malloc(resolution_x * resolution_y * sizeof(unsigned int));
    }
    // Rebuild the grid of complex numbers based on (new) center_x and (new) center_y.
    build_complex_grid();

    // Reset the amount of rendered iterations to 0. 
    rendered_iterations = 0;
}


void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (action == GLFW_PRESS)
    {
        switch (key) {
        case GLFW_KEY_D:
            if (debugging_enabled) {
                debugging_enabled = false;
            }
            else {
                debugging_enabled = true;
            }
            reset_render_objects();
            break;
        case GLFW_KEY_EQUAL: // zoom in, = is also +
            draw_radius *= 0.75; // zoom in
            reset_render_objects();
            break;
        case GLFW_KEY_MINUS: 
            draw_radius /= 0.75; // zoom out
            reset_render_objects();
            break;
        case GLFW_KEY_LEFT:
            printf("Changing center_x from: %f to: ", center_x);
            center_x -= 0.1 * draw_radius;
            printf("%f", center_x);
            reset_render_objects();
            break;
        case GLFW_KEY_RIGHT:
            printf("Changing center_x from: %f to: ", center_x);
            center_x += 0.1 * draw_radius;
            reset_render_objects();
            printf("%f", center_x);
            break;
        case GLFW_KEY_UP:
            center_y += 0.1 * draw_radius;
            reset_render_objects();
            break;
        case GLFW_KEY_DOWN:
            center_y -= 0.1 * draw_radius;
            reset_render_objects();
            break;
        case GLFW_KEY_LEFT_BRACKET:
            if (max_iterations > 2 && max_iterations < 10) {
                max_iterations--;
            }
            else if (max_iterations >= 10) {
                max_iterations *= 0.9;
            }
            printf("Max iterations now at: %d\n", max_iterations);
            if (incremental_iteration) {
                iterations_per_frame = incremental_iterations_per_frame;
            }
            else {
                iterations_per_frame = max_iterations;
            }
            reset_render_objects();
            break;
        case GLFW_KEY_RIGHT_BRACKET:
            if (max_iterations < 10) {
                max_iterations++;
            }
            else if (max_iterations >= 10){
                max_iterations /= 0.9;
            }
            printf("Max iterations now at: %d\n", max_iterations);
            if (incremental_iteration) {
                iterations_per_frame = incremental_iterations_per_frame;
            }
            else {
                iterations_per_frame = max_iterations;
            }
            reset_render_objects();
            break;
        case GLFW_KEY_I:
            if (incremental_iteration)
            {
                iterations_per_frame = incremental_iterations_per_frame;
                incremental_iteration = false;
            }
            else {
                iterations_per_frame = max_iterations;
                incremental_iteration = true;
            }
            break;
        case GLFW_KEY_ESCAPE:
            // Set the close flag of the window to TRUE so that the program exits:
            glfwSetWindowShouldClose(window, GL_TRUE);
            break;
        }

    }
}

void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{
    if (yoffset > 0) {
        draw_radius *= 0.75; // zoom in
        reset_render_objects();
    }
    else if (yoffset < 0) {
        draw_radius /= 0.75; // zoom out
        reset_render_objects();
    }
}

void allocate_memory() {
    if (cuda_device_available) {
        hipMallocManaged(&points_real, resolution_x * resolution_y * sizeof(double));
        hipMallocManaged(&points_imag, resolution_x * resolution_y * sizeof(double));
        hipMallocManaged(&iterated_points_real, resolution_x * resolution_y * sizeof(double));
        hipMallocManaged(&iterated_points_imag, resolution_x * resolution_y * sizeof(double));
        hipMallocManaged(&squared_absolute_values, resolution_x * resolution_y * sizeof(double));
        hipMallocManaged(&pixels_rgb, resolution_x * resolution_y * 3 * sizeof(unsigned char));
        hipMallocManaged(&iterationsArr, resolution_x * resolution_y * sizeof(unsigned int));
    }
    else if (!(cuda_device_available)) {
        points_real = (double *)malloc(resolution_x * resolution_y * sizeof(double));
        points_imag = (double*)malloc(resolution_x * resolution_y * sizeof(double));
        iterated_points_real = (double*)malloc(resolution_x * resolution_y * sizeof(double));
        iterated_points_imag = (double*)malloc(resolution_x * resolution_y * sizeof(double));
        squared_absolute_values = (double*)malloc(resolution_x * resolution_y * sizeof(double));
        pixels_rgb = (unsigned char*)malloc(resolution_x * resolution_y * 3 * sizeof(unsigned char));
        iterationsArr = (unsigned int*)malloc(resolution_x * resolution_y * sizeof(unsigned int));
    }
}

void free_the_pointers() {
    if (cuda_device_available) {
        hipFree(points_real);
        hipFree(points_imag);
        hipFree(iterated_points_real);
        hipFree(iterated_points_imag);
        hipFree(squared_absolute_values);
        hipFree(pixels_rgb);
        hipFree(iterationsArr);
    }
    else if (!(cuda_device_available)) {
        free(points_real);
        free(points_imag);
        free(iterated_points_real);
        free(iterated_points_imag);
        free(squared_absolute_values);
        free(pixels_rgb);
        free(iterationsArr);
    }
}

void setup_incremental_iterations() {
    if (incremental_iteration) {
        iterations_per_frame = incremental_iterations_per_frame;
    }
    else {
        iterations_per_frame = max_iterations;
    }
}

int main() {
    // Check for CUDA devices:
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0)
    {
        int cuda_device_id;
        hipGetDevice(&cuda_device_id);
        hipDeviceProp_t cuda_device_properties;
        hipGetDeviceProperties(&cuda_device_properties, cuda_device_id);
        cuda_device_available = true;
        printf("Using CUDA device: %s\n", cuda_device_properties.name);
        printf("cuda_num_blocks: %d\nblockSize: %d\n", cuda_num_blocks, cuda_block_size);
    } else {
        cuda_device_available = false;
        printf("No CUDA compatible devices found. Using CPU to compute images - performance will be limited.\n");
    }

    // Setup:
    allocate_memory();
    build_complex_grid();
    mandelbrot_iterate_and_color();
    setup_incremental_iterations();


    // Initialize the library 
    if (!glfwInit())
        return -1;

    // Create a windowed mode window and its OpenGL context 
    window = glfwCreateWindow(resolution_x, resolution_y, "Hello World", NULL, NULL);
    char* window_title = (char*)malloc(256);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    // Make the window's context current
    glfwMakeContextCurrent(window);
    glfwSetKeyCallback(window, key_callback);
    glfwSetScrollCallback(window, scroll_callback);

    // Loop until the window is closed
    while (!glfwWindowShouldClose(window))
    {
        if (debugging_enabled)
        {
            Sleep(500); // cap fps to 2
        }

        // Render here 
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);


        // TODO: make it so that the iterations increase for as long as the center and draw_radius are the same - up to the max of course
        if (rendered_iterations < max_iterations) {
            printf("Rendering %d iterations...\n", iterations_per_frame);
            mandelbrot_iterate_n_and_color(iterations_per_frame);
            rendered_iterations += iterations_per_frame;
            sprintf(window_title, "Max iterations: %d | points[0]: RE: %.32f IM: %.32f", max_iterations, points_real[0], points_imag[0]);
            glfwSetWindowTitle(window, window_title);
        }

        glDrawPixels(resolution_x, resolution_y, GL_RGB, GL_UNSIGNED_BYTE, pixels_rgb);

        // Swap front and back buffers 
        glfwSwapBuffers(window);

        // Poll for and process events 
        glfwPollEvents();
    }

    glfwTerminate();
    free_the_pointers();

	return 0;
}
